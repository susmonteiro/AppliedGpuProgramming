
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include <stdio.h>
#include <time.h>

#define NUM_ITER 20
#define count 1000000


__global__ void kernel(int* c, hiprandState *states){
    int id = threadIdx.x + blockDim.x * blockIdx.x;
	
	int seed = id; // different seed per thread
    hiprand_init(seed, id, 0, &states[id]);  // 	Initialize CURAND

    int inside = 0;
    for(int i = 0; i < count; i++) {
        float x = hiprand_uniform(&states[id]);
        float y = hiprand_uniform(&states[id]);
        if(sqrt(x*x + y*y) < 1) ++inside;
    }
    
    c[id] = inside;
    // printf("[%d] %d\n", id, c[id]);
    return;
}

int main() {
    hiprandState *dev_random;
    hipMalloc((void**)&dev_random, size_t(NUM_ITER) * sizeof(hiprandState));

    int* d_c = NULL;
    hipMalloc((void**)&d_c, size_t(NUM_ITER) * sizeof(int));
    
    // Time
    clock_t t;
    t = clock();

    kernel<<<(size_t(NUM_ITER) + 127) / 128, 128>>>(d_c, dev_random);
    hipDeviceSynchronize();


    int* out_gpu = (int*) malloc(size_t(NUM_ITER) * sizeof(int));
    memset(out_gpu, 0, size_t(NUM_ITER) * sizeof(int));
    hipMemcpy(out_gpu, d_c, size_t(NUM_ITER) * sizeof(int), hipMemcpyDeviceToHost);

    
    // Time
    t = clock() - t;
    printf("Time: %f\n", ((float)t)/CLOCKS_PER_SEC);

    int sum = 0;
    for(int i = 0; i < size_t(NUM_ITER); i++) {
        // printf("After [%d] %d\n", i, out_gpu[i]);
        sum += out_gpu[i];
    }

    float pi = 4 * ((float) sum / (count * size_t(NUM_ITER)));
    printf("PI=%f\n", pi);

    hipFree(d_c);
    hipFree(dev_random);
    free(out_gpu);

    return 0;
}
